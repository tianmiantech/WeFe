#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/


#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include <hip/hip_runtime.h>
#include "cgbn/cgbn.h"
#include "../utility/support.h"
#include <iostream>
#include <list>
#include <tuple>
#include <sstream>      //  istringstream、ostringstream和stringstream
#include <vector>
#include <bitset>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>


namespace py = pybind11;

using namespace std;

template<typename T>
string toString(const T& t){
    ostringstream oss;  //创建一个格式化输出流
    oss<<t;             //把值传递如流中
    return oss.str();  
}

template <class T>
int getLength(const T &arr){ //由于我们不知道T是什么，所以采用 pass by reference-to-const
  return sizeof(arr) / sizeof(arr[0]);
};

//一般方法，32位，逐步与1做与运算。=====================================
void Binarycout(uint32_t n)
{
  for (int i = 31; i >= 0; i--)
  {
    cout << ((n >> i) & 1);
  }
}

struct powmod_param_int {
    std::vector<uint32_t> x;
    std::vector<uint32_t> p;
    std::vector<uint32_t> modulus;

    powmod_param_int(vector<uint32_t> _x, vector<uint32_t> _p, vector<uint32_t> _modulus)
    {
      x = _x;
      p = _p;
      modulus = _modulus;
    }
};


// 数组转 vector, 逆序返回
std::vector<uint32_t> to_vector(uint32_t *arrHeight, uint32_t count){

  std::vector<uint32_t> int_vec(count);
  for(int index=0; index < count ; index++){
    int_vec[(count - 1) - index ] = arrHeight[index];
  }
  return int_vec;
}

std::vector<uint32_t> str_to_vector(std::string str, uint32_t bits){


  int count = bits / 32;
  vector<uint32_t> vec(count);
  for (int i = 0; i < count ; i++){
    // vec 正序
    cout << str.substr(i * 32,  32)  << endl;
    uint32_t value = stol(str.substr(i * 32, 32), nullptr, 2);
    vec[i] = value;
  }

  return vec;
}


template <uint32_t tpi, uint32_t bits, uint32_t window_bits>
class powm_params_t
{
public:
  // parameters used by the CGBN context
  static const uint32_t TPB = 0;           // get TPB from blockDim.x
  static const uint32_t MAX_ROTATION = 4;  // good default value
  static const uint32_t SHM_LIMIT = 0;     // no shared mem available
  static const bool CONSTANT_TIME = false; // constant time implementations aren't available yet

  // parameters used locally in the application
  static const uint32_t TPI = tpi;                 // threads per instance
  static const uint32_t BITS = bits;               // instance size
  static const uint32_t WINDOW_BITS = window_bits; // window size
};

//使用拓展欧几里得算法求e的模n的逆元d
bool getModInverse(const mpz_t e, const mpz_t n, mpz_t d)
{
  mpz_t a, b, c, c1, t, q, r;
  mpz_inits(a, b, c, c1, t, q, r, NULL);
  mpz_set(a, n);     //a=n;
  mpz_set(b, e);     //b=e;
  mpz_set_ui(c, 0);  //c=0
  mpz_set_ui(c1, 1); //c1=1
  mpz_tdiv_qr(q, r, a, b);
  while (mpz_cmp_ui(r, 0)) //r==0终止循环
  {
    mpz_mul(t, q, c1); //t=q*c1
    mpz_sub(t, c, t);  //t=c-q*c1

    mpz_set(c, c1); //c=c1  向后移动
    mpz_set(c1, t); //c1=t  向后移动

    mpz_set(a, b);           //a=b 除数变为被除数
    mpz_set(b, r);           //b=r 余数变为除数，开始下一轮
    mpz_tdiv_qr(q, r, a, b); //取下一个q
  }
  mpz_set(d, t); //将最后一轮的t赋值给d, d就是e的模n的逆元

  //保证返回正整数
  mpz_add(d, d, n);
  mpz_mod(d, d, n);

  mpz_clears(a, c, t, q, r, NULL);

  if (mpz_cmp_ui(b, 1))
  {
    mpz_clear(b);
    return false;
  }
  else
  {
    mpz_clear(b);
    return true;
  }
}

//蒙哥马利模乘，A=(A*B)mod n
void MontMult(mpz_t A, mpz_t B, const mpz_t n, int n_bit, const mpz_t IN)
{
  mpz_t T, T1, t0, b_32;
  mpz_inits(T, T1, t0, b_32, NULL);

  mpz_mul(T, A, B); //T=A*B
  mpz_set_ui(b_32, 0xFFFFFFFF);

  int t = n_bit >> 5;
  for (int i = 0; i < t; i++)
  {
    mpz_and(t0, T, b_32);
    mpz_mul(t0, IN, t0);
    mpz_and(t0, t0, b_32);

    //T1=T+n*t0
    mpz_mul(T1, n, t0);
    mpz_add(T1, T, T1);

    //T1>>32，T=T1
    mpz_tdiv_q_2exp(T1, T1, 32);
    mpz_set(T, T1);
  }

  if (mpz_cmp(T1, n) > 0)
  { //T1>n，A=T1-n
    mpz_sub(T1, T1, n);
    mpz_set(A, T1);
  }
  else
    mpz_set(A, T1);
  mpz_clears(T, T1, t0, b_32, NULL);
}

template <class params>
class powm_odd_t
{
public:
  static const uint32_t window_bits = params::WINDOW_BITS; // used a lot, give it an instance variable

  // define the instance structure
  typedef struct
  {
    cgbn_mem_t<params::BITS> x;
    cgbn_mem_t<params::BITS> power;
    cgbn_mem_t<params::BITS> modulus;
    cgbn_mem_t<params::BITS> result;
    // cgbn_mem_t<params::BITS> r_high;
  } instance_t;


  typedef cgbn_context_t<params::TPI, params> context_t;
  typedef cgbn_env_t<context_t, params::BITS> env_t;
  typedef typename env_t::cgbn_t bn_t;
  typedef typename env_t::cgbn_wide_t bn_wide_t;
  typedef typename env_t::cgbn_local_t bn_local_t;

  context_t _context;
  env_t _env;
  int32_t _instance;

  __device__ __forceinline__ powm_odd_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) : _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance)
  {
  }

  __device__ __forceinline__ void fixed_window_powm_odd(bn_t &result, const bn_t &x, const bn_t &power, const bn_t &modulus)
  {
    bn_t t;
    bn_local_t window[1 << window_bits];
    int32_t index, position, offset;
    uint32_t np0;

    // conmpute x^power mod modulus, using the fixed window algorithm
    // requires:  x<modulus,  modulus is odd

    // compute x^0 (in Montgomery space, this is just 2^BITS - modulus)
    cgbn_negate(_env, t, modulus);
    cgbn_store(_env, window + 0, t);

    // convert x into Montgomery space, store into window table
    np0 = cgbn_bn2mont(_env, result, x, modulus);
    cgbn_store(_env, window + 1, result);
    cgbn_set(_env, t, result);

// compute x^2, x^3, ... x^(2^window_bits-1), store into window table
#pragma nounroll
    for (index = 2; index < (1 << window_bits); index++)
    {
      cgbn_mont_mul(_env, result, result, t, modulus, np0);
      cgbn_store(_env, window + index, result);
    }

    // find leading high bit
    position = params::BITS - cgbn_clz(_env, power);

    // break the exponent into chunks, each window_bits in length
    // load the most significant non-zero exponent chunk
    offset = position % window_bits;
    if (offset == 0)
      position = position - window_bits;
    else
      position = position - offset;
    index = cgbn_extract_bits_ui32(_env, power, position, window_bits);
    cgbn_load(_env, result, window + index);

    // process the remaining exponent chunks
    while (position > 0)
    {
// square the result window_bits times
#pragma nounroll
      for (int sqr_count = 0; sqr_count < window_bits; sqr_count++)
        cgbn_mont_sqr(_env, result, result, modulus, np0);

      // multiply by next exponent chunk
      position = position - window_bits;
      index = cgbn_extract_bits_ui32(_env, power, position, window_bits);
      cgbn_load(_env, t, window + index);
      cgbn_mont_mul(_env, result, result, t, modulus, np0);
    }

    // we've processed the exponent now, convert back to normal space
    cgbn_mont2bn(_env, result, result, modulus, np0);
  }

  __device__ __forceinline__ void sliding_window_powm_odd(bn_t &result, const bn_t &x, const bn_t &power, const bn_t &modulus)
  {
    bn_t t, starts;
    int32_t index, position, leading;
    uint32_t mont_inv;
    bn_local_t odd_powers[1 << window_bits - 1];

    // conmpute x^power mod modulus, using Constant Length Non-Zero windows (CLNZ).
    // requires:  x<modulus,  modulus is odd

    // find the leading one in the power
    leading = params::BITS - 1 - cgbn_clz(_env, power);
    if (leading >= 0)
    {
      // convert x into Montgomery space, store in the odd powers table
      mont_inv = cgbn_bn2mont(_env, result, x, modulus);

      // compute t=x^2 mod modulus
      cgbn_mont_sqr(_env, t, result, modulus, mont_inv);

      // compute odd powers window table: x^1, x^3, x^5, ...
      cgbn_store(_env, odd_powers, result);
#pragma nounroll
      for (index = 1; index < (1 << window_bits - 1); index++)
      {
        cgbn_mont_mul(_env, result, result, t, modulus, mont_inv);
        cgbn_store(_env, odd_powers + index, result);
      }

      // starts contains an array of bits indicating the start of a window
      cgbn_set_ui32(_env, starts, 0);

      // organize p as a sequence of odd window indexes
      position = 0;
      while (true)
      {
        if (cgbn_extract_bits_ui32(_env, power, position, 1) == 0)
          position++;
        else
        {
          cgbn_insert_bits_ui32(_env, starts, starts, position, 1, 1);
          if (position + window_bits > leading)
            break;
          position = position + window_bits;
        }
      }

      // load first window.  Note, since the window index must be odd, we have to
      // divide it by two before indexing the window table.  Instead, we just don't
      // load the index LSB from power
      index = cgbn_extract_bits_ui32(_env, power, position + 1, window_bits - 1);
      cgbn_load(_env, result, odd_powers + index);
      position--;

      // Process remaining windows
      while (position >= 0)
      {
        cgbn_mont_sqr(_env, result, result, modulus, mont_inv);
        if (cgbn_extract_bits_ui32(_env, starts, position, 1) == 1)
        {
          // found a window, load the index
          index = cgbn_extract_bits_ui32(_env, power, position + 1, window_bits - 1);
          cgbn_load(_env, t, odd_powers + index);
          cgbn_mont_mul(_env, result, result, t, modulus, mont_inv);
        }
        position--;
      }

      // convert result from Montgomery space
      cgbn_mont2bn(_env, result, result, modulus, mont_inv);
    }
    else
    {
      // p=0, thus x^p mod modulus=1
      cgbn_set_ui32(_env, result, 1);
    }
  }


  __host__ static void verify_results(instance_t *instances, uint32_t count)
  {
    mpz_t x, p, m, computed, correct;

    mpz_init(x);
    mpz_init(p);
    mpz_init(m);
    mpz_init(computed);
    mpz_init(correct);

    for (int index = 0; index < count; index++)
    {
      to_mpz(x, instances[index].x._limbs, params::BITS / 32);
      to_mpz(p, instances[index].power._limbs, params::BITS / 32);
      to_mpz(m, instances[index].modulus._limbs, params::BITS / 32);
      to_mpz(computed, instances[index].result._limbs, params::BITS / 32);

      // 蒙哥马利算法 计算  ( a * b ) % N
      // mpz_powm(correct, x, p, m);
      // getModInverse(m, p, IN);

      // MontMult(x, p, m, 1024, IN);

      if (mpz_cmp(x, computed) != 0)
      {
        printf("gpu inverse kernel failed on instance %d\n", index);
        return;
      }
    }

    mpz_clear(x);
    mpz_clear(p);
    mpz_clear(m);
    mpz_clear(computed);
    mpz_clear(correct);

    printf("All results match\n");
  }

  __host__ static void str_to_limbs(uint32_t *x_list, vector<uint32_t> bignum, uint32_t count) {
    
    for(int index=0;index<count;index++){
        // 逆序放入数组
        x_list[(count - 1 )- index] = bignum[index];
    }
  }

      // transform to instances
      // __host__ static instance_t *to_instances(std::vector<powmod_param_int> arrs, uint32_t bits, uint32_t instance_count) {
      //   instance_t *instances=(instance_t *)malloc(sizeof(instance_t)*instance_count);

      //   for (int index=0; index < instance_count ; index++){
      //       str_to_limbs(instances[index].x._limbs, arrs[index].x , bits/32);
      //       str_to_limbs(instances[index].power._limbs, arrs[index].p , bits/32);
      //       str_to_limbs(instances[index].modulus._limbs,  arrs[index].modulus , bits/32);
      //   }
      //   return instances;
      // }

    __host__ static std::time_t getTimeStamp()
    {
        std::chrono::time_point<std::chrono::system_clock,std::chrono::milliseconds> tp = std::chrono::time_point_cast<std::chrono::milliseconds>(std::chrono::system_clock::now());//获取当前时间点
        std::time_t timestamp =  tp.time_since_epoch().count(); //计算距离1970-1-1,00:00的时间长度
        return timestamp;
    }


    __host__ static std::time_t getTimeStamp()
    {
        std::chrono::time_point<std::chrono::system_clock,std::chrono::milliseconds> tp = std::chrono::time_point_cast<std::chrono::milliseconds>(std::chrono::system_clock::now());//获取当前时间点
        std::time_t timestamp =  tp.time_since_epoch().count(); //计算距离1970-1-1,00:00的时间长度
        return timestamp;
    }


  __host__ static instance_t *to_instances(std::vector<tuple<py::bytes, py::bytes, py::bytes>> arrs, uint32_t bits, uint32_t instance_count)
  {
    instance_t *instances = (instance_t *)malloc(sizeof(instance_t) * instance_count);
    // cout << "copy start currentTime =  " << getTimeStamp() << endl;
    for (int index = 0; index < instance_count; index++)
    {
      char *x, *p, *modulus;

      Py_ssize_t len;
      PyBytes_AsStringAndSize(std::get<0>(arrs[index]).ptr(), &x, &len);
      PyBytes_AsStringAndSize(std::get<1>(arrs[index]).ptr(), &p, &len);
      PyBytes_AsStringAndSize(std::get<2>(arrs[index]).ptr(), &modulus, &len);

      memcpy(&(instances[index].x._limbs), x, len);
      memcpy(&(instances[index].power._limbs), p, len);
      memcpy(&(instances[index].modulus._limbs), modulus, len);
    }
    // cout << "copy end currentTime =  " << getTimeStamp() << endl;
    return instances;
  }

  __host__ static instance_t *to_instances_2(std::vector<py::bytes> arrs, py::bytes p_byte, py::bytes m_byte, uint32_t bits, uint32_t instance_count)
  {
    instance_t *instances = (instance_t *)malloc(sizeof(instance_t) * instance_count);

    for (int index = 0; index < instance_count; index++)
    {
      char *x, *p, *modulus;

      Py_ssize_t len;
      PyBytes_AsStringAndSize(arrs[index].ptr(), &x, &len);
      PyBytes_AsStringAndSize(p_byte.ptr(), &p, &len);
      PyBytes_AsStringAndSize(m_byte.ptr(), &modulus, &len);

      memcpy(&(instances[index].x._limbs), x, len);
      memcpy(&(instances[index].power._limbs), p, len);
      memcpy(&(instances[index].modulus._limbs), modulus, len);
    }
    // cout << "copy end currentTime =  " << getTimeStamp() << endl;
    return instances;
  }

  __host__ static instance_t *to_instances_2(std::vector<py::bytes> arrs, py::bytes p_byte, py::bytes m_byte, uint32_t bits, uint32_t instance_count)
  {
    instance_t *instances = (instance_t *)malloc(sizeof(instance_t) * instance_count);

    for (int index = 0; index < instance_count; index++)
    {
      char *x, *p, *modulus;

      Py_ssize_t len;
      PyBytes_AsStringAndSize(arrs[index].ptr(), &x, &len);
      PyBytes_AsStringAndSize(p_byte.ptr(), &p, &len);
      PyBytes_AsStringAndSize(m_byte.ptr(), &modulus, &len);

      memcpy(&(instances[index].x._limbs), x, len);
      memcpy(&(instances[index].power._limbs), p, len);
      memcpy(&(instances[index].modulus._limbs), modulus, len);

    }
    return instances;
  }


  // __host__ static std::vector<std::vector<uint32_t>> result_to_list(powm_odd_t<params>::instance_t *instances, uint32_t bits, uint32_t count){
      
  //     std::vector<std::vector<uint32_t>> result_list(count);
  //     for (int index = 0; index < count ; index++){
  //       // to_vector 方法逆序转化
  //       result_list[index] = to_vector(instances[index].result._limbs, bits/32);
  //     }
  //     return result_list;
  // }
  __host__ static std::vector<py::bytes> result_to_list(powm_odd_t<params>::instance_t *instances, uint32_t bits, uint32_t instance_count)
  {

    std::vector<py::bytes> result_list(instance_count);

    for (int i = 0;i < instance_count ; i++){
      py::bytes result = py::bytes((char *)instances[i].result._limbs, bits / 8);
      // cout << result << endl;
      result_list[i] = result;
    }
    return result_list;
  }
};


template <class params>
__global__ void kernel_powm_odd(cgbn_error_report_t *report, typename powm_odd_t<params>::instance_t *instances, uint32_t count)
{
  int32_t instance;

  // decode an instance number from the blockIdx and threadIdx
  instance = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (instance >= count)
    return;

  powm_odd_t<params> po(cgbn_report_monitor, report, instance);
  typename powm_odd_t<params>::bn_t r, x, p, m;

  // the loads and stores can go in the class, but it seems more natural to have them
  // here and to pass in and out bignums
  cgbn_load(po._env, x, &(instances[instance].x));
  cgbn_load(po._env, p, &(instances[instance].power));
  cgbn_load(po._env, m, &(instances[instance].modulus));

  // this can be either fixed_window_powm_odd or sliding_window_powm_odd.
  // when TPI<32, fixed window runs much faster because it is less divergent, so we use it here
  po.fixed_window_powm_odd(r, x, p, m);

  cgbn_store(po._env, &(instances[instance].result), r);
}


template <class params>
__global__ void kernel_mulm(cgbn_error_report_t *report, typename powm_odd_t<params>::instance_t *instances, uint32_t count)
{
  int32_t instance;

  // decode an instance number from the blockIdx and threadIdx
  instance = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (instance >= count)
    return;

  powm_odd_t<params> po(cgbn_report_monitor, report, instance);
  typename powm_odd_t<params>::bn_t  r, x, p, m, approx;
  typename powm_odd_t<params>::bn_wide_t w;

  // the loads and stores can go in the class, but it seems more natural to have them
  // here and to pass in and out bignums


  cgbn_load(po._env, x, &(instances[instance].x));
  cgbn_load(po._env, p, &(instances[instance].power));
  cgbn_load(po._env, m, &(instances[instance].modulus));

  uint32_t    clz_count;
  
  // compute the approximation of the inverse
  clz_count=cgbn_barrett_approximation(po._env, approx, m);
  
  // compute the wide product of a*b
  cgbn_mul_wide(po._env, w, x, p);
  
  // compute r=a*b mod d.  Pass the clz_count returned by the approx routine.
  cgbn_barrett_rem_wide(po._env, r, w, m, approx, clz_count);

  // cgbn_store(po._env, &(instances[instance].r_low), r_low );
  // cgbn_store(po._env, &(instances[instance].r_high), r_high );
  cgbn_store(po._env, &(instances[instance].result), r );

}


template <uint32_t tpi, uint32_t bits, uint32_t window_bits>
std::vector<py::bytes> powm(std::vector<tuple<py::bytes, py::bytes, py::bytes>> arrs, uint32_t instance_count)
{
  //   TPI             - threads per instance
  //   BITS            - number of bits per instance
  //   WINDOW_BITS     - number of bits to use for the windowed exponentiation

  typedef powm_params_t<tpi, bits, window_bits> params;
  // typedef powm_params_t<_tpi, _bits, _ window_bits> params;
  typedef typename powm_odd_t<params>::instance_t instance_t;

  instance_t *instances, *gpuInstances;
  cgbn_error_report_t *report;
  int32_t TPB = (params::TPB == 0) ? 128 : params::TPB; // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;           // IPB is instances per block

  instances = powm_odd_t<params>::to_instances(arrs, bits, instance_count);

  // printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));

  // cout << "copy to gpu start currentTime =  " << powm_odd_t<params>::getTimeStamp() << endl;
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t) * instance_count));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t) * instance_count, hipMemcpyHostToDevice));
  // cout << "copy to gpu end currentTime =  " << powm_odd_t<params>::getTimeStamp() << endl;

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  // printf("Running GPU kernel ...\n");

  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_powm_odd<params><<<(instance_count + IPB - 1) / IPB, TPB>>>(report, gpuInstances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
//   printf("Copying results back to CPU ...\n");
    // cout << "back to host start currentTime =  " << powm_odd_t<params>::getTimeStamp() << endl;
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t) * instance_count, hipMemcpyDeviceToHost));
    // cout << "back to host end currentTime =  " << powm_odd_t<params>::getTimeStamp() << endl;

  // printf("Verifying the results ...\n");
  // powm_odd_t<params>::verify_results(instances, instance_count);

  std::vector<py::bytes> gpu_result = powm_odd_t<params>::result_to_list(instances, bits, instance_count);

  // clean up
  free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));

  return gpu_result;
}



template <uint32_t tpi, uint32_t bits, uint32_t window_bits>
std::vector<py::bytes> powm_2(std::vector<py::bytes> arrs , py::bytes p, py::bytes m, uint32_t instance_count)
{
  //   TPI             - threads per instance
  //   BITS            - number of bits per instance
  //   WINDOW_BITS     - number of bits to use for the windowed exponentiation

  typedef powm_params_t<tpi, bits, window_bits> params;
  // typedef powm_params_t<_tpi, _bits, _ window_bits> params;
  typedef typename powm_odd_t<params>::instance_t instance_t;

  instance_t *instances, *gpuInstances;
  cgbn_error_report_t *report;
  int32_t TPB = (params::TPB == 0) ? 128 : params::TPB; // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;           // IPB is instances per block

  instances = powm_odd_t<params>::to_instances_2(arrs, p,m,bits, instance_count);

//   printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));

  // cout << "copy to gpu start currentTime =  " << powm_odd_t<params>::getTimeStamp() << endl;
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t) * instance_count));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t) * instance_count, hipMemcpyHostToDevice));
  // cout << "copy to gpu end currentTime =  " << powm_odd_t<params>::getTimeStamp() << endl;

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

//   printf("Running GPU kernel ...\n");

  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_powm_odd<params><<<(instance_count + IPB - 1) / IPB, TPB>>>(report, gpuInstances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
  // printf("Copying results back to CPU ...\n");
  // cout << "back to host start currentTime =  " << powm_odd_t<params>::getTimeStamp() << endl;
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t) * instance_count, hipMemcpyDeviceToHost));
  // cout << "back to host end currentTime =  " << powm_odd_t<params>::getTimeStamp() << endl;

  // printf("Verifying the results ...\n");
  // powm_odd_t<params>::verify_results(instances, instance_count);

  std::vector<py::bytes> gpu_result = powm_odd_t<params>::result_to_list(instances, bits, instance_count);

  // clean up
  free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));

  return gpu_result;
}



template <uint32_t tpi, uint32_t bits, uint32_t window_bits>
std::vector<py::bytes> powm_2(std::vector<py::bytes> arrs , py::bytes p, py::bytes m, uint32_t instance_count)
{
  //   TPI             - threads per instance
  //   BITS            - number of bits per instance
  //   WINDOW_BITS     - number of bits to use for the windowed exponentiation

  typedef powm_params_t<tpi, bits, window_bits> params;
  // typedef powm_params_t<_tpi, _bits, _ window_bits> params;
  typedef typename powm_odd_t<params>::instance_t instance_t;

  instance_t *instances, *gpuInstances;
  cgbn_error_report_t *report;
  int32_t TPB = (params::TPB == 0) ? 128 : params::TPB; // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;           // IPB is instances per block

  instances = powm_odd_t<params>::to_instances_2(arrs, p,m,bits, instance_count);

  // printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t) * instance_count));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t) * instance_count, hipMemcpyHostToDevice));

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  // printf("Running GPU kernel ...\n");

  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_powm_odd<params><<<(instance_count + IPB - 1) / IPB, TPB>>>(report, gpuInstances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
  // printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t) * instance_count, hipMemcpyDeviceToHost));

  // printf("Verifying the results ...\n");
  // powm_odd_t<params>::verify_results(instances, instance_count);

  std::vector<py::bytes> gpu_result = powm_odd_t<params>::result_to_list(instances, bits, instance_count);

  // clean up
  free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));

  return gpu_result;
}


template<uint32_t tpi, uint32_t bits, uint32_t window_bits>
std::vector<py::bytes> mulm(std::vector<tuple<py::bytes, py::bytes, py::bytes>> arrs, uint32_t instance_count) {
  //   TPI             - threads per instance
  //   BITS            - number of bits per instance
  //   WINDOW_BITS     - number of bits to use for the windowed exponentiation

  typedef powm_params_t<tpi, bits, window_bits> params;
  // typedef powm_params_t<_tpi, _bits, _ window_bits> params;
  typedef typename powm_odd_t<params>::instance_t instance_t;


  instance_t          *instances, *gpuInstances;
  cgbn_error_report_t *report;
  int32_t              TPB=(params::TPB==0) ? 128 : params::TPB;    // default threads per block to 128
  int32_t              TPI=params::TPI, IPB=TPB/TPI;                // IPB is instances per block

    
  instances = powm_odd_t<params>::to_instances(arrs, bits, instance_count);

  
  // printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)*instance_count));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t)*instance_count, hipMemcpyHostToDevice));
  
  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  
  // printf("Running GPU kernel ...\n");
  
  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_mulm<params><<<(instance_count+IPB-1)/IPB, TPB>>>(report, gpuInstances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);
    
  // copy the instances back from gpuMemory
  // printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t)*instance_count, hipMemcpyDeviceToHost));

  // std::vector<std::vector<uint32_t>> gpu_result = powm_odd_t<params>::result_to_list(instances,bits,instance_count);
  std::vector<py::bytes> gpu_result = powm_odd_t<params>::result_to_list(instances, bits, instance_count);


  // clean up
  free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));

  return gpu_result;
}


template<uint32_t tpi, uint32_t bits, uint32_t window_bits>
std::vector<py::bytes> mulm_2(std::vector<py::bytes> arrs , py::bytes p, py::bytes m, uint32_t instance_count) {
  //   TPI             - threads per instance
  //   BITS            - number of bits per instance
  //   WINDOW_BITS     - number of bits to use for the windowed exponentiation

  typedef powm_params_t<tpi, bits, window_bits> params;
  // typedef powm_params_t<_tpi, _bits, _ window_bits> params;
  typedef typename powm_odd_t<params>::instance_t instance_t;


  instance_t          *instances, *gpuInstances;
  cgbn_error_report_t *report;
  int32_t              TPB=(params::TPB==0) ? 128 : params::TPB;    // default threads per block to 128
  int32_t              TPI=params::TPI, IPB=TPB/TPI;                // IPB is instances per block


  instances = powm_odd_t<params>::to_instances_2(arrs, p, m, bits, instance_count);


//   printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)*instance_count));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t)*instance_count, hipMemcpyHostToDevice));

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

//   printf("Running GPU kernel ...\n");

  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_mulm<params><<<(instance_count+IPB-1)/IPB, TPB>>>(report, gpuInstances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
//   printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t)*instance_count, hipMemcpyDeviceToHost));

  // std::vector<std::vector<uint32_t>> gpu_result = powm_odd_t<params>::result_to_list(instances,bits,instance_count);
  std::vector<py::bytes> gpu_result = powm_odd_t<params>::result_to_list(instances, bits, instance_count);


  // clean up
  free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));

  return gpu_result;
}


template<uint32_t tpi, uint32_t bits, uint32_t window_bits>
std::vector<py::bytes> mulm_2(std::vector<py::bytes> arrs , py::bytes p, py::bytes m, uint32_t instance_count) {
  //   TPI             - threads per instance
  //   BITS            - number of bits per instance
  //   WINDOW_BITS     - number of bits to use for the windowed exponentiation

  typedef powm_params_t<tpi, bits, window_bits> params;
  // typedef powm_params_t<_tpi, _bits, _ window_bits> params;
  typedef typename powm_odd_t<params>::instance_t instance_t;


  instance_t          *instances, *gpuInstances;
  cgbn_error_report_t *report;
  int32_t              TPB=(params::TPB==0) ? 128 : params::TPB;    // default threads per block to 128
  int32_t              TPI=params::TPI, IPB=TPB/TPI;                // IPB is instances per block


  instances = powm_odd_t<params>::to_instances_2(arrs, p, m, bits, instance_count);


  // printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)*instance_count));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t)*instance_count, hipMemcpyHostToDevice));

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  // printf("Running GPU kernel ...\n");

  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_mulm<params><<<(instance_count+IPB-1)/IPB, TPB>>>(report, gpuInstances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
  // printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t)*instance_count, hipMemcpyDeviceToHost));

  // std::vector<std::vector<uint32_t>> gpu_result = powm_odd_t<params>::result_to_list(instances,bits,instance_count);
  std::vector<py::bytes> gpu_result = powm_odd_t<params>::result_to_list(instances, bits, instance_count);


  // clean up
  free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));

  return gpu_result;
}


PYBIND11_MODULE(gpu_lib, m) {

    // py::class_<powmod_param_int>(m, "powmod_param_int")
    //   .def(py::init<vector<uint32_t>, vector<uint32_t>, vector<uint32_t>>());
    m.def("mulm_2048", &mulm<8, 2048, 5>, py::return_value_policy::reference);
    m.def("mulm_1024", &mulm<8, 1024, 5>, py::return_value_policy::reference);

    m.def("powm_1024", &powm<8, 1024, 5>, py::return_value_policy::reference);
    m.def("powm_2048", &powm<8, 2048, 5>, py::return_value_policy::reference);
    m.def("powm_1024_2", &powm_2<8, 1024, 5>, py::return_value_policy::reference);
    m.def("powm_2048_2", &powm_2<8, 2048, 5>, py::return_value_policy::reference);
}
